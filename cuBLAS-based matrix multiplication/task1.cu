#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <time.h>
#include <chrono>
#include <stdlib.h>
#include <hipblas.h>
#include "mmul.h"

int main(int argc, char *argv[]){

    // 
    std::size_t n; std::size_t n_tests;
    n = std::atoi(argv[1]); 
    n_tests = std::atoi(argv[2]);

    using namespace std;
    int size = (n*n)*sizeof(float);
    float *A; float *B; float *C;

    hipMallocManaged(&A, size);
    hipMallocManaged(&B, size);
    hipMallocManaged(&C, size);
    srand(time(NULL));

    for(int i=0; i<n*n; i++){

        A[i]= (float)(rand()) / float(RAND_MAX) * (2) - 1;
        B[i]= (float)(rand()) / float(RAND_MAX) * (2) - 1;
    }
    
    float time = 0;
    
    for(int i = 0; i < n_tests; i++){
    // Create a handle for CUBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    mmul(handle,A,B,C,n);
    // Destroy the handle
    hipblasDestroy(handle);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
     
    // Get the elapsed time in milliseconds
    float ms = 0.f;
    hipEventElapsedTime(&ms, start, stop);
    time = time + ms;

    /*int j = 0;
    while(j < n*n){
        C[j] = 0;
        j++;
    }*/
    }
    
    cout << time/n_tests << endl;
    /*printf("A is  ");
    for(int i = 0; i < n*n; ++i){
        
        printf("%f ", A[i]);
    }
    printf ("\n");
    printf("B is  ");
     for(int i = 0; i < n*n; ++i){
        
        
        printf("%f ", B[i]);
    }
    printf ("\n");
    printf("C is  ");
     for(int i = 0; i < n*n; ++i){
        
        
        printf("%f ", C[i]);
    }*/
    return 0;
}
