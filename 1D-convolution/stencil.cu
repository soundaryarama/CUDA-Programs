#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "stencil.cuh"

__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R){

        int in = threadIdx.x + blockIdx.x * blockDim.x;
        
        extern __shared__ float sdata[]; //Declare shared memory
        int tid = threadIdx.x;

        int dR = (int)R;
                
        float* sharedM_mask = sdata;    
        //Copy mask from host to shared memory             
         if(tid < 2*dR+1){
                sharedM_mask[tid] = mask[tid];
        }

        float* sharedM_output = sharedM_mask+ 2*dR + 1;
        //Initialize output in shared memory to 0
        if(tid<n){
                sharedM_output[tid] = 0;
        }

        float* sharedM_image  =  sharedM_output + blockDim.x + dR;
        //Copy image from host to shared memory
         if(tid<n){
                sharedM_image[tid] = image[in];
        }

        //1D convolution boundary conditions
        if(tid<dR){
            if(in - dR > 0){
                sharedM_image[tid - dR] = image[in - dR];
            }
            else{
                sharedM_image[tid - dR] = 0;
            }
        }
        else if(blockDim.x - tid < dR){
            if(in + dR < n){
                sharedM_image[tid + dR] = image[in + dR]; 
            }
            else{
                sharedM_image[tid + dR] = 0;
            }
        }

        __syncthreads();

        for(int k = -dR; k <= dR; k++){
                sharedM_output[tid] += sharedM_image[tid + k] * sharedM_mask[k+dR];
        }


        output[in] = sharedM_output[tid]; 

}

__host__ void stencil(const float* image,
                      const float* mask,
                      float* output,
                      unsigned int n,
                      unsigned int R,
                      unsigned int threads_per_block){  
                                                        size_t blocks = (((n) + threads_per_block -1)/threads_per_block);
                                                        int sdata_size = ((2*R + threads_per_block) + (2*R+1) + (threads_per_block)) * sizeof(float);
                                                        stencil_kernel<<<blocks,threads_per_block, sdata_size>>>(image, mask, output, n, R);
                      }


            

