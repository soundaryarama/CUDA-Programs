#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <time.h>
#include <chrono>
#include <stdlib.h>
#include "scan.cuh"


int main(int argc, char *argv[]){

    // 
    unsigned int n; unsigned int threads_per_block;
    n = std::atoi(argv[1]); 
    threads_per_block = std::atoi(argv[2]);
    //n = 8; 
    //threads_per_block = 32;


    int size = (n)*sizeof(float);
    float *A; float *result;

    hipMallocManaged(&A, size);
    hipMallocManaged(&result, size);

    for(int i=0; i<n; i++){

         A[i]= (float)(rand()) / float(RAND_MAX) * (2) - 1;
        
    }
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    scan1(A,result,n,threads_per_block);
    
    hipEventRecord(stop);
    hipEventSynchronize(stop);
     
    // Get the elapsed time in milliseconds
    float ms = 0.f;
    hipEventElapsedTime(&ms, start, stop);

    printf("%f ", result[(n)-1]); printf("\n");
    printf("%f ", ms); printf("\n");
    

    
    return 0;
}
