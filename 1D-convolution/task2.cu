#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <time.h>
#include <chrono>
#include <stdlib.h>
#include "stencil.cuh"

int main(int argc, char *argv[]){

    // 
    std::size_t n; std::size_t threads; std::size_t R;
    n = std::atoi(argv[1]); 
    R = std::atoi(argv[2]);
    threads = std::atoi(argv[3]);
    //n = 4;

    srand(time(NULL));

    int size = (n)*sizeof(float);
    float* output = (float*) malloc((n)*sizeof(int));
    float* image = (float*) malloc((n)*sizeof(int));
    float* mask = (float*) malloc((2*R+1)*sizeof(int));

    for(int i=0; i<n; i++){

        image[i]= (float)(rand()) / float(RAND_MAX) * (2) - 1;
        
     }

    for(int i=0; i<(2*R+1); i++){

        mask[i]= (float)(rand()) / float(RAND_MAX) * (2) - 1;
    }

     float *dimage; float *dmask; float *doutput;

      // Allocate space for device
    hipMalloc((void **)&dimage, size);
    hipMalloc((void **)&dmask, size);
    hipMalloc((void **)&doutput, size);

    hipMemcpy(dimage, image, size, hipMemcpyHostToDevice);
    hipMemcpy(dmask, mask, size, hipMemcpyHostToDevice);

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    stencil(dimage,dmask,doutput,n, R, threads);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //Copy result from device to host
    hipMemcpy(output, doutput, size, hipMemcpyDeviceToHost);

    // Get the elapsed time in milliseconds
    float ms = 0.f;
    hipEventElapsedTime(&ms, start, stop);
 
    printf("%f ", output[n-1]); printf("\n");
    printf("%f ", ms); printf("\n");


return 0;
}
