#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "matmul.cuh"

__global__ void matmul_kernel(const float* A, const float* B, float* C, size_t n){
        
        int in = threadIdx.x + blockIdx.x * blockDim.x;
        if (in < n*n){
        float value = 0;
        
        for(int k = 0; k<n; k++){
                int rows  = in/n;
                int col = in%n;
                float first = A[rows*n+k];
                float second = B[col + k*n];
                value += first*second; 
                
        }
        C[in] = value;
        }
}

void matmul(const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block){
        size_t blocks = (((n*n) + threads_per_block -1)/threads_per_block);
        matmul_kernel<<<blocks,threads_per_block>>>(A, B, C, n);
}
