#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "reduce.cuh"

__global__ void reduce_kernel(float *g_idata, float *g_odata, unsigned int n){
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i   = (blockIdx.x*(2*blockDim.x)) + threadIdx.x;
    if (i >= n) {
        sdata[tid] = 0;
    } else {
        sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
    }

    __syncthreads();

    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
    if (tid < s) {
        sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
    }

    if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}

__host__ void reduce(float **input, float **output, unsigned int N,
                     unsigned int threads_per_block){
                        

    for (int l = N; l > 1; l = (l + threads_per_block - 1) / (2*threads_per_block)) {
        unsigned int block_num;
        if(l == N){
            block_num = ((l/2) + threads_per_block - 1)/(threads_per_block);
        }
        else{
            block_num = (l + threads_per_block - 1)/(threads_per_block);
        }
        reduce_kernel<<<block_num, threads_per_block, threads_per_block * sizeof(float) >>>(*input, *output, l);

        hipMemset(*input, 0, N * sizeof(float));

        hipMemcpy(*input, *output, block_num * sizeof(float), hipMemcpyDeviceToDevice);

        hipMemset(*output, 0, N * sizeof(float));

    }

    hipDeviceSynchronize();

}
