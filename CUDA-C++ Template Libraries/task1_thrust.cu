#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <time.h>
#include <chrono>
#include <stdlib.h>
#include <thrust/host_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>

int main(int argc, char *argv[]){

    std::size_t n;
    n = std::atoi(argv[1]); 
    thrust::host_vector<float> h_vec(n);
    srand(time(NULL));
    for(int i=0; i<n; i++){

         h_vec[i]= (float)(rand()) / float(RAND_MAX) * (2) - 1;
        
    }
    thrust::device_vector<float> d_vec = h_vec;

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    float result = thrust::reduce(d_vec.begin(), d_vec.end());
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Get the elapsed time in milliseconds
    float ms = 0.f;
    hipEventElapsedTime(&ms, start, stop);

    printf("%f ", result); printf("\n");
    printf("%f ", ms); printf("\n");
    
    //Check
    /*printf("A is  "); printf("\n"); float check = 0;
    for(int i = 0; i < n; ++i){
        
        printf("%f ", h_vec[i]); printf("\n");
        check = check + h_vec[i];

    }
    printf("Correct answer is  ");
    printf("%f ", check); printf("\n");*/
    
    return 0;
}





