#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <time.h>
#include <chrono>
#include <stdlib.h>
#include "reduce.cuh"

int main(int argc, char *argv[]){

    // 
    std::size_t n; std::size_t threads;
    n = std::atoi(argv[1]); 
    threads = std::atoi(argv[2]);
    //n = 6; 
    //threads = 1024;

    
    srand(time(NULL));
    // Check
   
    int size = (n)*sizeof(float);
    float* result = (float*) malloc((n)*sizeof(float));
    float* A = (float*) malloc((n)*sizeof(float));
    
     for(int i=0; i<n; i++){

        A[i]= (float)(rand()) / float(RAND_MAX) * (2) - 1;
        }
    
    
    float **dA; float **dresult;

    // Allocate space for device
    hipMalloc((void **)&dA, size);
    hipMalloc((void **)&dresult, size);

    hipMemcpy(dA, A, size, hipMemcpyHostToDevice);
    
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    reduce(dA,dresult,n,threads);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //Copy result from device to host
    hipMemcpy(result, dresult, size, hipMemcpyDeviceToHost);

    // Get the elapsed time in milliseconds
    float ms = 0.f;
    hipEventElapsedTime(&ms, start, stop);
 
    printf("%f ", result[0]); printf("\n");
    printf("%f ", ms); printf("\n");
    
    /*printf("A is  ");
    for(int i = 0; i < n; ++i){
        
        printf("%f ", A[i]); printf("\n");
    }

    printf("result is  ");
    for(int i = 0; i < n; ++i){
        
        printf("%f ", result[i]); printf("\n");
    }*/
    return 0;
}