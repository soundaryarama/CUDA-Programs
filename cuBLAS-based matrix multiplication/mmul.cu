#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "mmul.h"
#include <hipblas.h>

void mmul(hipblasHandle_t handle, const float* A, const float* B, float* C, int n){
    int lda=n,ldb=n,ldc=n;
    const float alf = 1;
    const float bet = 1;
    const float *alpha = &alf;
    const float *beta = &bet;


    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n,n,n,alpha, A, lda, B, ldb, beta, C, ldc);

    hipDeviceSynchronize();
}
//Referenced from
//https://solarianprogrammer.com/2012/05/31/matrix-multiplication-cuda-cublas-hiprand-thrust/
