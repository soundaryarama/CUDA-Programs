#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "reduce.cuh"

__global__ void reduce_kernel(float *g_idata, float *g_odata, unsigned int n){
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i   = blockIdx.x*(blockDim.x) + threadIdx.x;
    if (i >= n) {
        sdata[tid] = 0;
    } else {
        sdata[tid] = g_idata[i];
    }

    __syncthreads();

      for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
    if (tid < s) {
        sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
    }

    if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}

__host__ int reduce(const int* arr, unsigned int N, unsigned int threads_per_block) {
    int *g_idata;
    int *g_odata;
    using namespace std;
    hipMalloc(&g_idata, N * sizeof(int));

    hipMemcpy(g_idata, arr, N * sizeof(int), hipMemcpyHostToDevice);

    for (int l = N; l > 1; l = (l + threads_per_block - 1) / threads_per_block) {
        int block_num = (l + threads_per_block - 1) / threads_per_block;
        hipMalloc(&g_odata, block_num * sizeof(int));
        reduce_kernel<<<block_num, threads_per_block, threads_per_block * sizeof(int) >>>(g_idata, g_odata, l);


        hipMemcpy(g_idata, g_odata, block_num * sizeof(int), hipMemcpyDeviceToDevice);

    }

    hipDeviceSynchronize();
    
    hipFree(g_odata);
    hipFree(g_idata);

    int result;
    hipMemcpy(&result, g_odata, sizeof(int), hipMemcpyDeviceToHost);

    return result;
}
