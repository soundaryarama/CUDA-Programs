#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <time.h>
#include <chrono>
#include <stdlib.h>
#include <hipcub/hipcub.hpp>
#include <cub/util_allocator.cuh>
#include <cub/device/device_reduce.cuh>
#include "cub/util_debug.cuh"

using namespace hipcub;
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

int main(int argc, char** argv) {
    std::size_t n; n = atol(argv[1]);
    float *h_in = new float[n]; float result = 0;
    srand(time(NULL));
    for (int i = 0; i < n; i++) {
        h_in[i] = (float)(rand()) / float(RAND_MAX) * (2) - 1;
    }

    float* d_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_in, sizeof(float) * n));
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(float) * n, hipMemcpyHostToDevice));
    float* d_sum = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_sum, sizeof(float) * 1));
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms;
    hipEventElapsedTime(&ms, start, stop);

   
    CubDebugExit(hipMemcpy(&result, d_sum, sizeof(int) * 1, hipMemcpyDeviceToHost));

    printf("%f ", result); printf("\n");
    printf("%f ", ms); printf("\n");

    /*printf("A is  "); printf("\n"); float check = 0;
    for(int i = 0; i < n; ++i){
        
        printf("%f ", h_in[i]); printf("\n");
        check = check + h_in[i];

    }
    printf("Correct answer is  ");
    printf("%f ", check); printf("\n");*/
    // Cleanup
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_sum) CubDebugExit(g_allocator.DeviceFree(d_sum));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
    return 0;

}