#include "hip/hip_runtime.h"
#include "scan.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void scan_kernel(const float* input, float* output, int n) {
    extern volatile __shared__  float temp[]; // allocated on invocation
    float f = 0; 
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    // Copying data from global to shared memory
    if (index < n) {
        temp[tid] = input[index];
    }
    else {
        temp[tid] = 0;
    }
    __syncthreads();

    for(int step = 1; step <= blockDim.x; step *= 2) 
    {
        if(tid >= step)
        {
            f = temp[tid-step];
            temp[tid] += f;
        }
        __syncthreads();
    }
    __syncthreads();
    if (index < n)
        output[index] = temp[tid]; //output[index] = 
    
}

__global__ void extra(const float* input, float* output, unsigned int num_blocks, unsigned int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;
    
    for(int step = 1; step <= num_blocks; step *= 2) 
    {
        unsigned int start_block_idx = blockIdx.x * blockDim.x - (step - 1) * blockDim.x - 1;
        if(bid >= step && index < n)
        {
            output[index] += output[start_block_idx];
        }
        __syncthreads();
    }
    
}


__host__ void scan(const float* input, float* output, unsigned int n, unsigned int threads_per_block){
    unsigned int blocks = ((((n) + threads_per_block -1)/threads_per_block));
    unsigned int sdata_size = 2*threads_per_block;
    scan_kernel<<<blocks,threads_per_block, sdata_size>>>(input,output, n);
    //unsigned int n_blocks = ((((n) + threads_per_block -1)/threads_per_block));
    
        //scan_kernel<<<i,threads_per_block, sdata_size>>>(*input,output, n, threads_per_block);
    extra <<< blocks, threads_per_block >>> (input,output,blocks, n);
    
}