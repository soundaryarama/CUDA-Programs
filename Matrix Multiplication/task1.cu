#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <time.h>
#include <chrono>
#include <stdlib.h>
#include "matmul.cuh"

int main(int argc, char *argv[]){

    // 
    std::size_t n; std::size_t threads;
    n = std::atoi(argv[1]); 
    threads = std::atoi(argv[2]);
    //n = 4;
    
    //float A[n*n];
	//float B[n*n]; 
    //float result[n*n];
    srand(time(NULL));
    // Check
    /*
    float A[n*n] = {5,2,6,1,0,6,2,0,3,8,1,4,1,8,5,6};
    float B[n*n] = {7,5,8,0,1,8,2,6,9,4,3,8,5,3,7,9};*/

    int size = (n*n)*sizeof(float);
    float* result = (float*) malloc((n*n)*sizeof(int));
    float* A = (float*) malloc((n*n)*sizeof(int));
    float* B = (float*) malloc((n*n)*sizeof(int));

     for(int i=0; i<n*n; i++){

        A[i]= (float)(rand()) / float(RAND_MAX) * (2) - 1;
        B[i]= (float)(rand()) / float(RAND_MAX) * (2) - 1;
     }

     float *dA; float *dB; float *dC;

    // Allocate space for device
    hipMalloc((void **)&dA, size);
    hipMalloc((void **)&dB, size);
    hipMalloc((void **)&dC, size);

    hipMemcpy(dA, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, size, hipMemcpyHostToDevice);

    

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    matmul(dA,dB,dC,n,threads);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    //Copy result from device to host
    hipMemcpy(result, dC, size, hipMemcpyDeviceToHost);

    // Get the elapsed time in milliseconds
    float ms = 0.f;
    hipEventElapsedTime(&ms, start, stop);
 
    printf("%f ", result[(n*n)-1]); printf("\n");
    printf("%f ", ms); printf("\n");
   
}
